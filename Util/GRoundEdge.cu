#include "hip/hip_runtime.h"
#include "CUtilInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace MotionCor2::Util;

static __global__ void mGRoundEdge(float* gfImg, int iSizeX,
	float* gfMaskCent, float* gfMaskSize, float fScale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= iSizeX) return;
	int i = blockIdx.y * iSizeX + x;
	//------------------------------------
	float fX = 2 * fabsf(x - gfMaskCent[0]) / gfMaskSize[0];
	float fY = 2 * fabsf(blockIdx.y - gfMaskCent[1]) / gfMaskSize[1];
	float fR = sqrtf(fX * fX + fY * fY);
	if(fR >= 1.0f)
	{	gfImg[i] = 0.0f;
		return;
	}
	//-------------
	fR = 0.5f * (1 - cosf(3.1415926f * fR));
	fR = 1.0f - powf(fR, 3.0f);
	gfImg[i] = (gfImg[i] * fR) * fScale;
}


GRoundEdge::GRoundEdge(void)
{
	m_fScale = 1.0f;
}

GRoundEdge::~GRoundEdge(void)
{
}

void GRoundEdge::SetMask(float* pfCent, float* pfSize)
{
	m_afMaskCent[0] = pfCent[0];
	m_afMaskCent[1] = pfCent[1];
	m_afMaskSize[0] = pfSize[0];
	m_afMaskSize[1] = pfSize[1];
}

void GRoundEdge::SetScale(float fScale)
{
	m_fScale = fScale;
}

void GRoundEdge::DoIt(float* gfImg, int* piImgSize)
{
	size_t tBytes = 0;
	hipMemcpyKind aH2D = hipMemcpyHostToDevice;
	//-------------------------------------------
	float* gfMaskCent = 0L;
	tBytes = sizeof(float) * 2;
	hipMalloc(&gfMaskCent, tBytes);
	hipMemcpy(gfMaskCent, m_afMaskCent, tBytes, aH2D);
	//-------------------------------------------------
	float* gfMaskSize = 0L;
	hipMalloc(&gfMaskSize, tBytes);
	hipMemcpy(gfMaskSize, m_afMaskSize, tBytes, aH2D);
	//-------------------------------------------------
	dim3 aBlockDim(512, 1);
	int iGridX = piImgSize[0] / aBlockDim.x + 1;
	dim3 aGridDim(iGridX, piImgSize[1]);
	//----------------------------------
	mGRoundEdge<<<aGridDim, aBlockDim>>>(gfImg, piImgSize[0],
		gfMaskCent, gfMaskSize, m_fScale);
	//----------------------------------------
	if(gfMaskCent != 0L) hipFree(gfMaskCent);
	if(gfMaskSize != 0L) hipFree(gfMaskSize);
}
